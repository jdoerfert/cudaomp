
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// RUN: %clang -### -target x86_64-linux-gnu -nocudalib -ccc-print-bindings -fgpu-rdc \
// RUN:        -foffload-new-driver --offload-arch=sm_35 --offload-arch=sm_70 %s 2>&1 \
// RUN: | FileCheck -check-prefix CHECK %s

// CHECK: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT:.+]]"], output: "[[PTX_SM_35:.+]]"
// CHECK: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_35]]"], output: "[[CUBIN_SM_35:.+]]"
// CHECK: "nvptx64-nvidia-cuda" - "NVPTX::Linker", inputs: ["[[CUBIN_SM_35]]", "[[PTX_SM_35]]"], output: "[[FATBIN_SM_35:.+]]"
// CHECK: "nvptx64-nvidia-cuda" - "clang", inputs: ["[[INPUT]]"], output: "[[PTX_SM_70:.+]]"
// CHECK: "nvptx64-nvidia-cuda" - "NVPTX::Assembler", inputs: ["[[PTX_SM_70:.+]]"], output: "[[CUBIN_SM_70:.+]]"
// CHECK: "nvptx64-nvidia-cuda" - "NVPTX::Linker", inputs: ["[[CUBIN_SM_70]]", "[[PTX_SM_70:.+]]"], output: "[[FATBIN_SM_70:.+]]"
// CHECK: "x86_64-unknown-linux-gnu" - "clang", inputs: ["[[INPUT]]", "[[FATBIN_SM_35]]", "[[FATBIN_SM_70]]"], output: "[[HOST_OBJ:.+]]"
// CHECK: "x86_64-unknown-linux-gnu" - "Offload::Linker", inputs: ["[[HOST_OBJ]]"], output: "a.out"
